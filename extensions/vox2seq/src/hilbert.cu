#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
namespace cg = cooperative_groups;

#include "hilbert.h"


// Expands a 10-bit integer into 30 bits by inserting 2 zeros after each bit.
static __device__ uint32_t expandBits(uint32_t v)
{
    v = (v * 0x00010001u) & 0xFF0000FFu;
    v = (v * 0x00000101u) & 0x0F00F00Fu;
    v = (v * 0x00000011u) & 0xC30C30C3u;
    v = (v * 0x00000005u) & 0x49249249u;
    return v;
}


// Removes 2 zeros after each bit in a 30-bit integer.
static __device__ uint32_t extractBits(uint32_t v)
{
    v = v & 0x49249249;
    v = (v ^ (v >>  2)) & 0x030C30C3u;
    v = (v ^ (v >>  4)) & 0x0300F00Fu;
    v = (v ^ (v >>  8)) & 0x030000FFu;
    v = (v ^ (v >> 16)) & 0x000003FFu;
    return v;
}


__global__ void hilbert_encode_cuda(
    size_t N,
    const uint32_t* x,
    const uint32_t* y,
    const uint32_t* z,
    uint32_t* codes
) {
    size_t thread_id = cg::this_grid().thread_rank();
    if (thread_id >= N) return;

    uint32_t point[3] = {x[thread_id], y[thread_id], z[thread_id]};

    uint32_t m = 1 << 9, q, p, t;

    // Inverse undo excess work
    q = m;
    while (q > 1) {
        p = q - 1;
        for (int i = 0; i < 3; i++) {
            if (point[i] & q) {
                point[0] ^= p;  // invert
            } else {
                t = (point[0] ^ point[i]) & p;
                point[0] ^= t;
                point[i] ^= t;
            }
        }
        q >>= 1;
    }

    // Gray encode
    for (int i = 1; i < 3; i++) {
        point[i] ^= point[i - 1];
    }
    t = 0;
    q = m;
    while (q > 1) {
        if (point[2] & q) {
            t ^= q - 1;
        }
        q >>= 1;
    }
    for (int i = 0; i < 3; i++) {
        point[i] ^= t;
    }

    // Convert to 3D Hilbert code
    uint32_t xx = expandBits(point[0]);
    uint32_t yy = expandBits(point[1]);
    uint32_t zz = expandBits(point[2]);

    codes[thread_id] = xx * 4 + yy * 2 + zz;
}


__global__ void hilbert_decode_cuda(
    size_t N,
    const uint32_t* codes,
    uint32_t* x,
    uint32_t* y,
    uint32_t* z
) {
    size_t thread_id = cg::this_grid().thread_rank();
    if (thread_id >= N) return;

    uint32_t point[3];
    point[0] = extractBits(codes[thread_id] >> 2);
    point[1] = extractBits(codes[thread_id] >> 1);
    point[2] = extractBits(codes[thread_id]);

    uint32_t m = 2 << 9, q, p, t;

    // Gray decode by H ^ (H/2)
    t = point[2] >> 1;
    for (int i = 2; i > 0; i--) {
        point[i] ^= point[i - 1];
    }
    point[0] ^= t;

    // Undo excess work
    q = 2;
    while (q != m) {
        p = q - 1;
        for (int i = 2; i >= 0; i--) {
            if (point[i] & q) {
                point[0] ^= p;
            } else {
                t = (point[0] ^ point[i]) & p;
                point[0] ^= t;
                point[i] ^= t;
            }
        }
        q <<= 1;
    }

    x[thread_id] = point[0];
    y[thread_id] = point[1];
    z[thread_id] = point[2];
}
