#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
namespace cg = cooperative_groups;

#include "z_order.h"


// Expands a 10-bit integer into 30 bits by inserting 2 zeros after each bit.
static __device__ uint32_t expandBits(uint32_t v)
{
    v = (v * 0x00010001u) & 0xFF0000FFu;
    v = (v * 0x00000101u) & 0x0F00F00Fu;
    v = (v * 0x00000011u) & 0xC30C30C3u;
    v = (v * 0x00000005u) & 0x49249249u;
    return v;
}


// Removes 2 zeros after each bit in a 30-bit integer.
static __device__ uint32_t extractBits(uint32_t v)
{
    v = v & 0x49249249;
    v = (v ^ (v >>  2)) & 0x030C30C3u;
    v = (v ^ (v >>  4)) & 0x0300F00Fu;
    v = (v ^ (v >>  8)) & 0x030000FFu;
    v = (v ^ (v >> 16)) & 0x000003FFu;
    return v;
}


__global__ void z_order_encode_cuda(
    size_t N,
    const uint32_t* x,
    const uint32_t* y,
    const uint32_t* z,
    uint32_t* codes
) {
    size_t thread_id = cg::this_grid().thread_rank();
	if (thread_id >= N) return;

    uint32_t xx = expandBits(x[thread_id]);
    uint32_t yy = expandBits(y[thread_id]);
    uint32_t zz = expandBits(z[thread_id]);

    codes[thread_id] = xx * 4 + yy * 2 + zz;
}


__global__ void z_order_decode_cuda(
    size_t N,
    const uint32_t* codes,
    uint32_t* x,
    uint32_t* y,
    uint32_t* z
) {
    size_t thread_id = cg::this_grid().thread_rank();
    if (thread_id >= N) return;

    x[thread_id] = extractBits(codes[thread_id] >> 2);
    y[thread_id] = extractBits(codes[thread_id] >> 1);
    z[thread_id] = extractBits(codes[thread_id]);
}
